/*
5. Error Handling and Asynchronous Operations
Example: Error checking after kernel launches.
*/
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void dummyKernel() {
    // Simulate a simple task
}

int main() {
    // Launch kernel
    dummyKernel<<<1, 1>>>();

    // Check for errors
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
    }

    // Synchronize device
    hipDeviceSynchronize();

    // Check for post-synchronization errors
    err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA Synchronization Error: " << hipGetErrorString(err) << std::endl;
    }

    return 0;
}
