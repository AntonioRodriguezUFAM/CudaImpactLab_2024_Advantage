/*
3. Asynchronous Operations
Example: Asynchronous memory transfers with CUDA streams.
*/
#include <hip/hip_runtime.h>
#include <iostream>

int main() {
    int n = 1000000;
    float *h_data = (float*)malloc(n * sizeof(float));
    float *d_data;
    hipStream_t stream;

    // Initialize host data
    for (int i = 0; i < n; i++) {
        h_data[i] = static_cast<float>(i);
    }

    hipMalloc((void**)&d_data, n * sizeof(float));
    hipStreamCreate(&stream);

    // Asynchronous data transfer
    hipMemcpyAsync(d_data, h_data, n * sizeof(float), hipMemcpyHostToDevice, stream);

    // Perform operations (can be kernel calls in the same stream)
    // ...

    // Synchronize stream to ensure all operations are complete
    hipStreamSynchronize(stream);

    // Cleanup
    hipStreamDestroy(stream);
    hipFree(d_data);
    free(h_data);

    return 0;
}
