
/*
2. Parallel Vector Addition Using Threads and Blocks
Example: Vector addition kernel with blocks and threads.
*/

#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void vectorAdd(int *a, int *b, int *c, int n) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < n) {
        c[index] = a[index] + b[index];
    }
}

#define N 1024
#define THREADS_PER_BLOCK 256

int main() {
    int size = N * sizeof(int);
    int *a, *b, *c, *d_a, *d_b, *d_c;

    // Allocate host memory
    a = (int*)malloc(size);
    b = (int*)malloc(size);
    c = (int*)malloc(size);

    // Initialize vectors
    for (int i = 0; i < N; i++) {
        a[i] = i;
        b[i] = i * 2;
    }

    // Allocate device memory
    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_c, size);

    // Copy data to device
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    // Launch kernel
    int numBlocks = (N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    vectorAdd<<<numBlocks, THREADS_PER_BLOCK>>>(d_a, d_b, d_c, N);

    // Copy result back to host
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    // Free memory
    hipFree(d_a); hipFree(d_b); hipFree(d_c);
    free(a); free(b); free(c);

    return 0;
}
