/*

2. Error Handling in CUDA
Example: Kernel launch with error handling.
*/
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void simpleKernel() {
    // Simple dummy kernel
}

int main() {
    simpleKernel<<<1, 1>>>();
    hipError_t err = hipGetLastError();

    if (err != hipSuccess) {
        std::cerr << "Kernel launch error: " << hipGetErrorString(err) << std::endl;
    }

    hipDeviceSynchronize();
    err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "Post-synchronization error: " << hipGetErrorString(err) << std::endl;
    }

    return 0;
}
