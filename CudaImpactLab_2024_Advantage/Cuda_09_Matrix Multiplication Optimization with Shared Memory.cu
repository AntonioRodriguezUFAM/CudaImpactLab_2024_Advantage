/*
4. Matrix Multiplication Optimization with Shared Memory
Example: Matrix multiplication using shared memory for performance improvement.
*/
#include <stdio.h>
#include <hip/hip_runtime.h>

#define TILE_SIZE 16

__global__ void matMulShared(float *A, float *B, float *C, int N) {
    __shared__ float As[TILE_SIZE][TILE_SIZE];
    __shared__ float Bs[TILE_SIZE][TILE_SIZE];

    int row = blockIdx.y * TILE_SIZE + threadIdx.y;
    int col = blockIdx.x * TILE_SIZE + threadIdx.x;
    float sum = 0;

    for (int i = 0; i < (N + TILE_SIZE - 1) / TILE_SIZE; i++) {
        if (i * TILE_SIZE + threadIdx.x < N && row < N) {
            As[threadIdx.y][threadIdx.x] = A[row * N + i * TILE_SIZE + threadIdx.x];
        } else {
            As[threadIdx.y][threadIdx.x] = 0.0;
        }
        if (i * TILE_SIZE + threadIdx.y < N && col < N) {
            Bs[threadIdx.y][threadIdx.x] = B[(i * TILE_SIZE + threadIdx.y) * N + col];
        } else {
            Bs[threadIdx.y][threadIdx.x] = 0.0;
        }

        __syncthreads();

        for (int j = 0; j < TILE_SIZE; j++) {
            sum += As[threadIdx.y][j] * Bs[j][threadIdx.x];
        }
        __syncthreads();
    }

    if (row < N && col < N) {
        C[row * N + col] = sum;
    }
}

int main() {
    int N = 512;
    size_t size = N * N * sizeof(float);
    float *h_A, *h_B, *h_C, *d_A, *d_B, *d_C;

    // Allocate host memory
    h_A = (float*)malloc(size);
    h_B = (float*)malloc(size);
    h_C = (float*)malloc(size);

    // Initialize matrices
    for (int i = 0; i < N * N; i++) {
        h_A[i] = 1.0f;
        h_B[i] = 1.0f;
    }

    // Allocate device memory
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    // Copy data to device
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Launch kernel
    dim3 dimBlock(TILE_SIZE, TILE_SIZE);
    dim3 dimGrid((N + TILE_SIZE - 1) / TILE_SIZE, (N + TILE_SIZE - 1) / TILE_SIZE);
    matMulShared<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, N);

    // Copy result back to host
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Cleanup
    hipFree(d_A); hipFree(d_B); hipFree(d_C);
    free(h_A); free(h_B); free(h_C);

    return 0;
}
