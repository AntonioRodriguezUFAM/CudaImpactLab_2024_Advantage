/*
3. Matrix Multiplication Using CUDA Kernels
Example: Basic CUDA matrix multiplication.

*/

#include <stdio.h>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 16

__global__ void matrixMul(float *A, float *B, float *C, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float Cvalue = 0;

    if (row < N && col < N) {
        for (int k = 0; k < N; ++k) {
            Cvalue += A[row * N + k] * B[k * N + col];
        }
        C[row * N + col] = Cvalue;
    }
}

int main() {
    int N = 512; // Matrix size (N x N)
    size_t size = N * N * sizeof(float);
    float *h_A, *h_B, *h_C, *d_A, *d_B, *d_C;

    // Allocate host memory
    h_A = (float*)malloc(size);
    h_B = (float*)malloc(size);
    h_C = (float*)malloc(size);

    // Initialize matrices
    for (int i = 0; i < N * N; i++) {
        h_A[i] = 1.0f;
        h_B[i] = 1.0f;
    }

    // Allocate device memory
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    // Copy matrices to device
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Launch kernel with 2D grid and block dimensions
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid((N + dimBlock.x - 1) / dimBlock.x, (N + dimBlock.y - 1) / dimBlock.y);
    matrixMul<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, N);

    // Copy result back to host
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Free memory
    hipFree(d_A); hipFree(d_B); hipFree(d_C);
    free(h_A); free(h_B); free(h_C);

    return 0;
}
